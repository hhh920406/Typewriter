#include "hip/hip_runtime.h"
#include "reference_calc.cpp"
#include "utils.h"

__global__ 
void reduceMinKernel(float *d_out, const float* d_in)
{
    extern __shared__ float sdata[];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    sdata[tid] = d_in[idx];
    __syncthreads();

    for (unsigned int s = blockDim.x >> 1; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            if (sdata[tid] > sdata[tid + s])
            {
                sdata[tid] = sdata[tid + s];
            }
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        d_out[blockIdx.x] = sdata[0];
    }
}

__global__ 
void reduceMaxKernel(float *d_out, const float* d_in)
{
    extern __shared__ float sdata[];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    sdata[tid] = d_in[idx];
    __syncthreads();

    for (unsigned int s = blockDim.x >> 1; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            if (sdata[tid] < sdata[tid + s])
            {
                sdata[tid] = sdata[tid + s];
            }
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        d_out[blockIdx.x] = sdata[0];
    }
}

void reduce(float &min_logLum,
            float &max_logLum,
            float *d_out, 
            float *d_intermediate, 
            const float *d_in, 
            int size)
{
    int threads, blocks;

    threads = 1024;
    blocks = (size - 1) / 1024 + 1;
    reduceMinKernel<<<blocks, threads, threads * sizeof(float)>>>(d_intermediate, d_in);
    threads = blocks;
    blocks = 1;
    reduceMinKernel<<<blocks, threads, threads * sizeof(float)>>>(d_out, d_intermediate);
    hipMemcpy(&min_logLum, d_out, sizeof(float), hipMemcpyDeviceToHost);

    threads = 1024;
    blocks = (size - 1) / 1024 + 1;
    reduceMaxKernel<<<blocks, threads, threads * sizeof(float)>>>(d_intermediate, d_in);
    threads = blocks;
    blocks = 1;
    reduceMaxKernel<<<blocks, threads, threads * sizeof(float)>>>(d_out, d_intermediate);
    max_logLum = d_out[0];
    hipMemcpy(&max_logLum, d_out, sizeof(float), hipMemcpyDeviceToHost);
}

__global__
void histKernel(unsigned int *d_out, 
                const float *d_in, 
                float min_logLum, 
                float lumRange, 
                const int numBins)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int bin = (d_in[idx] - min_logLum) / lumRange * numBins;
    atomicAdd(&(d_out[bin]), 1);
}

void genHist(unsigned int *d_out, 
             const float *d_in, 
             size_t size,
             float min_logLum, 
             float max_logLum, 
             const int numBins)
{
    int threads = 1024;
    int blocks = (size - 1) / 1024 + 1;
    //histKernel<<<blocks, threads>>>(d_out, d_in, min_logLum, max_logLum - min_logLum, numBins);
}

void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
  //TODO
  /*Here are the steps you need to implement
    1) find the minimum and maximum value in the input logLuminance channel
       store in min_logLum and max_logLum
    2) subtract them to find the range
    3) generate a histogram of all the values in the logLuminance channel using
       the formula: bin = (lum[i] - lumMin) / lumRange * numBins
    4) Perform an exclusive scan (prefix sum) on the histogram to get
       the cumulative distribution of luminance values (this should go in the
       incoming d_cdf pointer which already has been allocated for you)       */
    
    float *d_out;
    float *d_intermediate;
    size_t size = numRows * numCols;
    hipMalloc((void**)&d_out, sizeof(float));
    hipMalloc((void**)&d_intermediate, sizeof(float) * size);
    reduce(min_logLum, max_logLum, d_out, d_intermediate, d_logLuminance, size);
    hipDeviceSynchronize(); 
    checkCudaErrors(hipGetLastError());
    unsigned int *d_hist;
    hipMalloc((void**)&d_hist, sizeof(unsigned int) * numBins);
    genHist(d_hist, d_logLuminance, size, min_logLum, max_logLum, numBins);
}

