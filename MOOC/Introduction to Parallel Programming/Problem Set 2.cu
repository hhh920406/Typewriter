#include "hip/hip_runtime.h"
#include "reference_calc.cpp"
#include "utils.h"

__global__
void gaussian_blur(const unsigned char* const inputChannel,
                   unsigned char* const outputChannel,
                   int numRows, int numCols,
                   const float* const filter, const int filterWidth)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= numCols || y >= numRows)
    {
        return;
    }
    int idx = y * numCols + x;
    int halfFilterWidth = filterWidth >> 1;
    float sum = 0.0;
    for (int i = 0; i < filterWidth; ++i)
    {
        for (int j = 0; j < filterWidth; ++j)
        {
            int tx = x + i - halfFilterWidth;
            int ty = y + j - halfFilterWidth;
            if (tx < 0) tx = 0;
            if (tx >= numCols) tx = numCols - 1;
            if (ty < 0) ty = 0;
            if (ty >= numRows) ty = numRows - 1;
            int tIdx = ty * numCols + tx;
            int fIdx = j * filterWidth + i;
            sum += float(inputChannel[tIdx]) * filter[fIdx];
        }
    }
    outputChannel[idx] = char(sum);
}

__global__
void separateChannels(const uchar4* const inputImageRGBA,
                      int numRows,
                      int numCols,
                      unsigned char* const redChannel,
                      unsigned char* const greenChannel,
                      unsigned char* const blueChannel)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= numCols || y >= numRows)
    {
        return;
    }
    int idx = y * numCols + x;
    redChannel[idx] = inputImageRGBA[idx].x;
    greenChannel[idx] = inputImageRGBA[idx].y;
    blueChannel[idx] = inputImageRGBA[idx].z;
}

__global__
void recombineChannels(const unsigned char* const redChannel,
                       const unsigned char* const greenChannel,
                       const unsigned char* const blueChannel,
                       uchar4* const outputImageRGBA,
                       int numRows,
                       int numCols)
{
    const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                          blockIdx.y * blockDim.y + threadIdx.y);
    const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;
    if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
        return;
    unsigned char red   = redChannel[thread_1D_pos];
    unsigned char green = greenChannel[thread_1D_pos];
    unsigned char blue  = blueChannel[thread_1D_pos];
    uchar4 outputPixel = make_uchar4(red, green, blue, 255);
    outputImageRGBA[thread_1D_pos] = outputPixel;
}

unsigned char *d_red, *d_green, *d_blue;
float         *d_filter;

void allocateMemoryAndCopyToGPU(const size_t numRowsImage, const size_t numColsImage,
                                const float* const h_filter, const size_t filterWidth)
{
    checkCudaErrors(hipMalloc(&d_red,   sizeof(unsigned char) * numRowsImage * numColsImage));
    checkCudaErrors(hipMalloc(&d_green, sizeof(unsigned char) * numRowsImage * numColsImage));
    checkCudaErrors(hipMalloc(&d_blue,  sizeof(unsigned char) * numRowsImage * numColsImage));
    checkCudaErrors(hipMalloc(&d_filter, sizeof(float) * filterWidth * filterWidth));
    checkCudaErrors(hipMemcpy(d_filter, h_filter, sizeof(float) * filterWidth * filterWidth, hipMemcpyHostToDevice));
}

void your_gaussian_blur(const uchar4 * const h_inputImageRGBA, uchar4 * const d_inputImageRGBA,
                        uchar4* const d_outputImageRGBA, const size_t numRows, const size_t numCols,
                        unsigned char *d_redBlurred, 
                        unsigned char *d_greenBlurred, 
                        unsigned char *d_blueBlurred,
                        const int filterWidth)
{
    const dim3 gridSize((numCols - 1) / 16 + 1, (numRows - 1) / 16 + 1, 1);
    const dim3 blockSize(16, 16, 1);

    separateChannels<<<gridSize, blockSize>>>(d_inputImageRGBA, numRows, numCols, d_red, d_green, d_blue);
    hipDeviceSynchronize(); 
    checkCudaErrors(hipGetLastError());

    gaussian_blur<<<gridSize, blockSize>>>(d_red, d_redBlurred, numRows, numCols, d_filter, filterWidth);
    gaussian_blur<<<gridSize, blockSize>>>(d_green, d_greenBlurred, numRows, numCols, d_filter, filterWidth);
    gaussian_blur<<<gridSize, blockSize>>>(d_blue, d_blueBlurred, numRows, numCols, d_filter,filterWidth);
    hipDeviceSynchronize(); 
    checkCudaErrors(hipGetLastError());

    recombineChannels<<<gridSize, blockSize>>>(d_redBlurred, d_greenBlurred, d_blueBlurred, d_outputImageRGBA, numRows, numCols);
    hipDeviceSynchronize(); 
    checkCudaErrors(hipGetLastError());
}

void cleanup() 
{
    checkCudaErrors(hipFree(d_red));
    checkCudaErrors(hipFree(d_green));
    checkCudaErrors(hipFree(d_blue));
    checkCudaErrors(hipFree(d_filter));
}

