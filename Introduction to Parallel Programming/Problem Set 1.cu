#include "hip/hip_runtime.h"
#include "reference_calc.cpp"
#include "utils.h"
#include <stdio.h>

__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
    int x = blockIdx.x;
    int y = threadIdx.x;
    int idx = x * numCols + y;
    float r = rgbaImage[idx].x;
    float g = rgbaImage[idx].y;
    float b = rgbaImage[idx].z;
    float i = 0.299f * r + 0.587f * g + 0.114f * b;
    greyImage[idx] = i;
}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
    const dim3 gridSize(numRows, 1, 1);
    const dim3 blockSize(numCols, 1, 1);
    rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
    hipDeviceSynchronize(); 
    checkCudaErrors(hipGetLastError());
}

